#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>

__global__ void merge_images(uint8_t* dest, uint8_t* src1, int width1, 
	int height1, uint8_t* src2, int width2, int height2){

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width1 && y < height1) {
		int index1 = y * width1 + x;
		int index2 = (y * height2 / height1) * width2 + (x * width2 / width1);

	/*
		dest[index1 * 4 + 0] = (src1[index1 * 4 + 0] + src2[index2 * 4 + 0]) / 2;
		dest[index1 * 4 + 1] = (src1[index1 * 4 + 1] + src2[index2 * 4 + 1]) / 2;
		dest[index1 * 4 + 2] = (src1[index1 * 4 + 2] + src2[index2 * 4 + 2]) / 2;
		dest[index1 * 4 + 3] = (src1[index1 * 4 + 3] + src2[index2 * 4 + 3]) / 2;
	*/
		dest[index1 * 3 + 0] = (src1[index1 * 3 + 0] + src2[index2 * 3 + 0]) / 2;
		dest[index1 * 3 + 1] = (src1[index1 * 3 + 1] + src2[index2 * 3 + 1]) / 2;
		dest[index1 * 3 + 2] = (src1[index1 * 3 + 2] + src2[index2 * 3 + 2]) / 2;
	}
}

int main(int argc, char* argv[]){

	if(argc != 3){
		fprintf(stderr, "Usage: ./merge inputFile1.png inputFile2.png");
		exit(1);
	}

	int width1, height1, channels1;

	//uint8_t *image1 = stbi_load(argv[1], &width1, &height1, &channels1, STBI_rgb_alpha);
	uint8_t *image1 = stbi_load(argv[1], &width1, &height1, &channels1, STBI_rgb);

	int width2, height2, channels2;

	//uint8_t *image2 = stbi_load(argv[2], &width2, &height2, &channels2, STBI_rgb_alpha);
	uint8_t *image2 = stbi_load(argv[2], &width2, &height2, &channels2, STBI_rgb);

	// Allocate memory on the device
	uint8_t *dev_dest, *dev_src1, *dev_src2;

	/*
	hipMalloc((void**)&dev_dest, width1 * height1 * 4);
	hipMalloc((void**)&dev_src1, width1 * height1 * 4);
	hipMalloc((void**)&dev_src2, width2 * height2 * 4);
	*/
	hipMalloc((void**)&dev_dest, width1 * height1 * 3);
	hipMalloc((void**)&dev_src1, width1 * height1 * 3);
	hipMalloc((void**)&dev_src2, width2 * height2 * 4);

	// Copy the input images to the device
	//hipMemcpy(dev_src1, image1, width1 * height1 * 4, hipMemcpyHostToDevice);
	//hipMemcpy(dev_src2, image2, width2 * height2 * 4, hipMemcpyHostToDevice);
	hipMemcpy(dev_src1, image1, width1 * height1 * 3, hipMemcpyHostToDevice);
	hipMemcpy(dev_src2, image2, width2 * height2 * 3, hipMemcpyHostToDevice);

	// Define block and grid sizes
	dim3 blockSize(16, 16);
	dim3 gridSize((width1 + blockSize.x - 1) / blockSize.x, (height1 + blockSize.y - 1) / blockSize.y);

	// Call the kernel to merge the images
	merge_images<<<gridSize, blockSize>>>(dev_dest, dev_src1, width1, height1, dev_src2, width2, height2);

	// Copy the result back to the host
	//uint8_t *result = (uint8_t*)malloc(width1 * height1 * 4);
	//hipMemcpy(result, dev_dest, width1 * height1 * 4, hipMemcpyDeviceToHost);
	uint8_t *result = (uint8_t*)malloc(width1 * height1 * 3);
	hipMemcpy(result, dev_dest, width1 * height1 * 3, hipMemcpyDeviceToHost);

	// Save the result to disk
	//stbi_write_png("result.png", width1, height1, 4, result, width1 * 4);
	stbi_write_png("result.jpg", width1, height1, 3, result, width1 * 3);

	// Free memory on the host and device
	free(result);
	stbi_image_free(image1);
	stbi_image_free(image2);
	hipFree(dev_dest);
	hipFree(dev_src1);
	hipFree(dev_src2);
	return 0;
}