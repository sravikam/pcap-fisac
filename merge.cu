#include "hip/hip_runtime.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define CHANNELS 3

typedef unsigned char uchar;

__global__ void merge_images(uchar* dest, uchar* src1, int width1, 
	int height1, uchar* src2, int width2, int height2){

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width1 && y < height1) {
		int index1 = (y * width1 + x)*CHANNELS;
		int index2 = ((y * height2 / height1) * width2 + (x * width2 / width1))*CHANNELS;

		for(int j=0; j<CHANNELS; ++j){
			dest[index1 + j] = (src1[index1 + j] + src2[index2 + j]) / 2;
		}
	}
}

// Returns length if valid or 0 if invalid
int isValidFilename(const char* filename){
	const char* invalidChars = "\\/:*?\"<>|";
	int len = strlen(filename);
	for (int i = 0; i < len; ++i) {
		if (strchr(invalidChars, filename[i]) != NULL) {
			return 0;
		}
	}
	return len;
}

int main(int argc, char* argv[]){

	if(argc < 3 || argc > 4){
		fprintf(stderr, "Usage: ./merge inputFile1.png inputFile2.png [outputFile]");
		exit(1);
	}

	char* outputFileName;
	
	if(argc == 4){
		if(strrchr(argv[3], '.')){
			fprintf(stderr, "Error: Output file should not be given an extension\n");
			exit(1);
		}
		int value = isValidFilename(argv[3]);
		if(value){
			outputFileName = (char*)calloc(value+5, sizeof*outputFileName);
			strcpy(outputFileName, argv[3]);
			strcat(outputFileName, ".png");
		} else{
			fprintf(stderr, "Error: Invalid output file name");
		}
	} else{
		outputFileName = (char*)calloc(11, sizeof*outputFileName);
		strcpy(outputFileName, "result.png");
	}
	

	int width1, height1;
	uchar *image1 = stbi_load(argv[1], &width1, &height1, NULL, STBI_rgb);
	if(!image1){
		fprintf(stderr, "Error: %s\n", stbi_failure_reason());
		exit(1);
	}

	int width2, height2;
	uchar *image2 = stbi_load(argv[2], &width2, &height2, NULL, STBI_rgb);
	if(!image2){
		fprintf(stderr, "Error: %s\n", stbi_failure_reason());
		exit(1);
	}

	// Allocate memory on the device
	uchar *dev_dest, *dev_src1, *dev_src2;

	hipMalloc((void**)&dev_dest, width1 * height1 * CHANNELS);
	hipMalloc((void**)&dev_src1, width1 * height1 * CHANNELS);
	hipMalloc((void**)&dev_src2, width2 * height2 * CHANNELS);

	// Copy the input images to the device
	hipMemcpy(dev_src1, image1, width1 * height1 * CHANNELS, hipMemcpyHostToDevice);
	hipMemcpy(dev_src2, image2, width2 * height2 * CHANNELS, hipMemcpyHostToDevice);

	// Define block and grid sizes
	dim3 blockSize(16, 16);
	dim3 gridSize((width1 + blockSize.x - 1) / blockSize.x, (height1 + blockSize.y - 1) / blockSize.y);

	// Call the kernel to merge the images
	merge_images<<<gridSize, blockSize>>>(dev_dest, dev_src1, width1, height1, dev_src2, width2, height2);

	// Copy the result back to the host
	uchar *result = (uchar*)malloc(width1 * height1 * CHANNELS);
	hipMemcpy(result, dev_dest, width1 * height1 * CHANNELS, hipMemcpyDeviceToHost);

	// Save the result to disk
	stbi_write_png(outputFileName, width1, height1, CHANNELS, result, width1 * CHANNELS);

	// Free memory on the host and device
	free(result);
	free(image1);
	free(image2);
	free(outputFileName);
	hipFree(dev_dest);
	hipFree(dev_src1);
	hipFree(dev_src2);
	return 0;
}